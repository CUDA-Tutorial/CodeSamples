#include <hip/hip_runtime_api.h>
#include <iostream>
#include "../../shared/include/test_scheduling.cuh"

int main()
{
    std::cout << "==== Sample 05 ====\n";
    std::cout << "==== Independent Thread Scheduling ====\n" << std::endl;
    /*
     This code will launch a particular test kernel.
     It will launch 4 threads in total.
     The program code is structured such that each
     thread enters one of 4 possible branches and then
     atomically increments a GPU variable N times:
    
                .---- N operations by Thread 0
           ----X
         /      '---- N operations by Thread 1
    ----X
         \      .---- N operations by Thread 2
           ----X
                '---- N operations by Thread 3
    
     Each thread will document consecutive ranges of 
     values it observed for the incremented variable.  
     Basically, this will give us an idea how threads
     take turns running in this branching scenario.
    
     Expected output: Many smaller observed value ranges!
     Independent thread scheduling is free to switch 
     between branches to keep the compute units of the 
     GPU busy with work. Thus, threads can alternate 
     taking turns to complete their N steps each.
    
     Disclaimer: behavior is dependent on scheduling at
     runtime. Results may vary.
     */

    const int N = 128;
    run2NestedBranchesForNSteps(N);
    return 0;
}
