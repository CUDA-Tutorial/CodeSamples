
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <algorithm>
#include <thread>
#include <atomic>
#include <mutex>
#include <condition_variable>

// A simple kernel function to keep threads busy for a while
__global__ void busy()
{
	const int start = clock();
	while ((clock() - start) < 1'000'000'000);
	printf("I'm awake!\n");
}

int main()
{
	std::cout << "==== Sample 08 - Streams ====\n" << std::endl;
	/*
	 Expected output: "I'm awake!\n" x 4 x KERNEL_CALLS + 4

	 If you watch the output carefully or analyze the execution of 
	 this program with NVIDIA Nsight Systems, it should show that the 
	 first group of kernels run consecutively, while the second and 
	 third group run in parallel. 
	 
	 Finally, there should be two kernels running sequentially,
	 followed by two kernels running in parallel.
	*/

	constexpr unsigned int KERNEL_CALLS = 2;

	std::cout << "Running sequential launches" << std::endl;
	// Launch the same kernel several times in a row
	for (unsigned int i = 0; i < KERNEL_CALLS; i++)
		busy<<<1, 1>>>();
	// Synchronize before continuing to get clear separation in Nsight
	hipDeviceSynchronize();

	std::cout << "\nRunning launches in streams" << std::endl;
	// Allocate one stream for each kernel to be launched
	hipStream_t streams[KERNEL_CALLS];
	for (hipStream_t& s : streams)
	{
		// Create stream and launch kernel into it
		hipStreamCreate(&s);
		busy<<<1, 1, 0, s>>>();
	}
	// Destroy all streams (implicitly waits until each has finished)
	for (hipStream_t& s : streams)
		hipStreamDestroy(s);
	hipDeviceSynchronize();

	/*
	If we don't specify a stream, then the kernel is launched into the default 
	stream. Also, many operations like cudaDeviceSynchronize and 
	cudaStreamSynchronize are submitted to the default stream. Usually, only a 
	single default stream is defined per application, meaning that if you don't 
	specify streams, you will not be able to benefit from kernels running 
	concurrently. Hence, any elaborate CUDA application should be using streams. 
	
	However, if the task can be cleanly separated into CPU threads, there is another 
	option: using per-thread default streams. Each thread will use its own default
	stream if we pass the built-in value cudaStreamPerThread as the stream to use.
	Kernels can then run concurrently on the GPU by creating multiple CPU threads.
	Alternatively, you may set the compiler option "--default-stream per-thread". 
	This way, CPU threads will use separate default streams if none are specified.
	*/
	std::cout << "\nRunning threads with different default streams" << std::endl;

	// Create mutex, condition variable and counter for communication
	std::mutex mutex;
	std::condition_variable cv;
	unsigned int kernelsLaunched = 0;
	// Allocate sufficient number of threads
	std::thread threads[KERNEL_CALLS];
	// Create a separate thread for each kernel call (task)
	for (std::thread& t : threads)
	{
		t = std::thread([&mutex, &cv, &kernelsLaunched] {
			// Launch kernel to thread's default stream
			busy<<<1, 1, 0, hipStreamPerThread>>>();
			/*
			 Make sure all kernels are submitted before synchronizing,
			 because cudaStreamSynchronize goes into the default 0 stream:
			 busy<1> -> sync<0>(1) -> busy<2> -> sync<0>(2)... serializes.
			 busy<1> -> busy<2> -> sync<0>(1) -> sync<0>(2)... parallelizes.
			*/
			std::unique_lock<std::mutex> lock(mutex);
			++kernelsLaunched;
			cv.wait(lock, [&kernelsLaunched] { return kernelsLaunched == KERNEL_CALLS; });
			cv.notify_all();
			// Synchronize to wait for printf output
			hipStreamSynchronize(hipStreamPerThread);
		});
	}
	// Wait for all threads to finish launching their kernels in individual streams
	std::for_each(threads, threads + KERNEL_CALLS, [](std::thread& t) {t.join(); });

	/*
	By default, custom created streams will implicitly synchronize with the 
	default stream. Consider, e.g., a kernel A running in a custom stream, 
	followed by a kernel B in the default stream. If we use cudaStreamCreate
	as above, then A will end before B starts. Alternatively, we may create 
	custom streams with the flag cudaStreamNonBlocking. In this case, the 
	custom stream will not synchronize with the default stream anymore. 
	*/
	hipStream_t customRegular, customNonblocking;
	hipStreamCreate(&customRegular);
	hipStreamCreateWithFlags(&customNonblocking, hipStreamNonBlocking);

	auto testAB = [](const char* kind, hipStream_t stream) {
		std::cout << "\nLaunching A (custom) -> B (default) with " << kind << " custom stream" << std::endl;
		busy<<<1, 1, 0, stream>>>();
		busy<<<1, 1>>>();
		hipDeviceSynchronize();
	};

	testAB("regular", customRegular);
	testAB("non-blocking", customNonblocking);

	// Clean up generated streams
	hipStreamDestroy(customRegular);
	hipStreamDestroy(customNonblocking);

	return 0;
}
