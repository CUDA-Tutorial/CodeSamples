
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>

__device__ int dVal = 42;
__device__ int dOut;

// Very simple kernel that updates a variable
__global__ void CopyVal(int* val)
{
	// Simulating a little work
	const int start = clock();
	while ((clock() - start) < 1'000'000);

	// Update a global value
	dOut = *val;
}

void checkForErrors()
{
	// Catch errors that can be detected without synchronization
	hipError_t err;
	err = hipGetLastError();
	if (err == hipSuccess)
		std::cout << "cudaGetLastError() before sync found no error" << std::endl;
	else
		std::cout << "cudaGetLastError() before sync found error: " << hipGetErrorName(err) << ", CLEARS ERROR" << std::endl;

	// Catch errors that require explicit synchronization
	err = hipDeviceSynchronize();
	if (err == hipSuccess)
		std::cout << "cudaDeviceSynchronize() found no error" << std::endl;
	else
		std::cout << "cudaDeviceSynchronize() found error: " << hipGetErrorName(err) << ", KEEPS ERROR" << std::endl;

	// If errors were found via synchronization, cudaGetLastError clears them
	err = hipGetLastError();
	if (err == hipSuccess)
		std::cout << "cudaGetLastError() after sync found no error" << std::endl;
	else
		std::cout << "cudaGetLastError() after sync found error: " << hipGetErrorName(err) << ", CLEARS ERROR" << std::endl;

	std::cout << std::endl;
}

#define PRINT_RUN_CHECK(S)		\
std::cout << #S << std::endl;	\
S;								\
checkForErrors();

int main()
{
	std::cout << "==== Sample 10 - Error Handling ====\n" << std::endl;
	/*
	 Many functions in the CUDA API return error codes that indicate
	 that something has gone wrong. However, this error is not 
	 necessarily caused by the function that returns it. Kernels and
	 asynchronous memcopies e.g. are launched immediately and may only
	 encounter errors after the return value is observed on the CPU. 
	 Such errors can be detected at some later point, for instance by
	 a synchronous function like cudaMemcpy or cudaDeviceSynchronize,
	 or by cudaGetLastError after a synchronization. To ensure that 
	 every CUDA call worked without error, we would have to sacrifice 
	 concurrency and asynchronicity. Hence, error checking is, in practice,
	 rather opportunistic and happens e.g. at runtime when an algorithm
	 is synchronized anyway or when we debug misbehaving code. The error
	 checking in this code is thus not practical and only serves to 
	 illustrate how different mechanisms detect and affect previous errors. 

	 Expected output:

		(CopyVal<<<1, 1>>>(validDAddress))
		cudaGetLastError() before sync found no error
		cudaDeviceSynchronize() found no error
		cudaGetLastError() after sync found no error

		(CopyVal<<<1, (1<<16)>>>(validDAddress))
		cudaGetLastError() before sync found error: cudaErrorInvalidConfiguration, CLEARS ERROR
		cudaDeviceSynchronize() found no error
		cudaGetLastError() after sync found no error

		(CopyVal<<<1, 1>>>(nullptr))
		cudaGetLastError() before sync found no error
		cudaDeviceSynchronize() found error: cudaErrorIllegalAddress, KEEPS ERROR
		cudaGetLastError() after sync found error: cudaErrorIllegalAddress, CLEARS ERROR

		cudaErrorInvalidPc: invalid program counter
	*/

	int* validDAddress;
	// A function may return an error code - should check those for success
	hipError_t err = hipGetSymbolAddress((void**)&validDAddress, HIP_SYMBOL(dVal));

	if (err != hipSuccess)
		// If an error occurred, identify it with cudaGetErrorName and react!
		std::cout << hipGetErrorName(err) << std::endl;
	// Alternatively, you may peek at the last error to see if the program is ok
	err = hipPeekAtLastError();
	// Getting the last error effectively resets it. Useful after reacting to it
	err = hipGetLastError();

	/* 
	Launching a kernel with proper configuration and parameters.
	If the system is set up correctly, this should succeed.
	*/
	PRINT_RUN_CHECK((CopyVal<<<1, 1>>>(validDAddress)));

	/* 
	Launching a kernel with bigger block than possible.
	cudaGetLastError() can catch SOME errors without synchronizing!
	*/
	PRINT_RUN_CHECK((CopyVal<<<1, (1<<16)>>>(validDAddress)));

	/*
	Launching a kernel with invalid address - error occurs after launch
	cudaGetLastError() alone may miss this without synchronization.
	*/
	PRINT_RUN_CHECK((CopyVal<<<1, 1>>>(nullptr)));

	// For any kind of error, CUDA also provides a more verbose description.
	std::cout << hipGetErrorName(cudaErrorInvalidPc) << ": " << hipGetErrorString(cudaErrorInvalidPc) << std::endl;
}