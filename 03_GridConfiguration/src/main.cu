#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>

__global__ void PrintIDs()
{
    // Use built-in variables blockIdx and threadIdx
    const auto tID = threadIdx;
    const auto bID = blockIdx;
    printf("Block Id: %d,%d - Thread Id: %d,%d\n", bID.x, bID.y, tID.x, tID.y);
}

int main()
{
    std::cout << "==== Sample 03 - Grid Configurations ====\n" << std::endl;
    /*
    Expected output:
    Block IDs and Thread IDs for two separate grids
    */

    std::cout << "Small grid: \n";
    // Configure the grid and block dimensions via built-in struct dim3 (X,Y,Z)
    const dim3 gridSize_small{ 1, 1, 1 };
    const dim3 blockSize_small{ 4, 4, 1 };

    // Launch kernel with custom grid
    PrintIDs<<<gridSize_small, blockSize_small>>>();

    // Need to synchronize here to have the GPU and CPU printouts in the correct order
    hipDeviceSynchronize();

    std::cout << "\nLarger grid: \n";
    const dim3 gridSize_large{ 2, 2, 1 };
    const dim3 blockSize_large{ 16, 16, 1 };
    PrintIDs<<<gridSize_large, blockSize_large >>>();
    hipDeviceSynchronize();

    return 0;
}

/*
Exercises:
1) Launch a 1D grid with 2D blocks (e.g., 4 x (4 x 4))
2) Launch a 2D grid with 2D blocks (e.g., (4 x 4) x (4 x 4))
3) Launch a grid where either the grid or block are 3D
4) Can you find out what the limits are to how many blocks/threads you can launch?
*/