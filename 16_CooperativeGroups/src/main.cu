#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "../../shared/include/generate_random.h"

// Shortening cooperative groups namespace for convenience
namespace cg = cooperative_groups;

// We keep the result of the reduction in managed memory
__managed__ float mResult;

template <unsigned int BLOCK_SIZE>
__global__ void reduceGroups(const float* __restrict input, int N)
{
    // Can conveniently obtain groups for grid and block
    auto block = cg::this_thread_block();
    unsigned int gId = cg::this_grid().thread_rank();

    __shared__ float data[BLOCK_SIZE];
    data[block.thread_rank()] = (gId < N ? (input[gId] + input[gId + N / 2]) : 0);

    for (int s = blockDim.x / 2; s > 16; s /= 2)
    {
        // Rather than selecting explicit sync functions, groups offer sync()
        block.sync();
        if (block.thread_rank() < s)
            data[block.thread_rank()] += data[block.thread_rank() + s];
    }

    // Splitting blocks into warp groups is cleaner than checking threadIdx
    auto warp = cg::tiled_partition<32>(block);
    if (warp.meta_group_rank() == 0)
    {
        // Reduction primitives - will be hardware-accelerated on CC 8.0+
        float v = cg::reduce(warp, data[warp.thread_rank()], cg::plus<float>());
        if (warp.thread_rank() == 0)
            atomicAdd(&mResult, v);
    }
}

void ReduceWithGroups()
{
    constexpr unsigned int BLOCK_SIZE = 256, N = 1'000'000;

    std::cout << "Producing random inputs...\n" << std::endl;
    // Generate some random numbers to reduce
    std::vector<float> vals;
    float* dValsPtr;
    prepareRandomNumbersCPUGPU(N, vals, &dValsPtr);
    // Prepare grid configuration for input and used reduction technique
    const dim3 blockDim = { BLOCK_SIZE, 1, 1 };
    const dim3 gridDim = { (N / 2 + BLOCK_SIZE) / BLOCK_SIZE, 1, 1 };

    // Events for measuring run time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Setting managed result variable
    mResult = 0;
    hipEventRecord(start);
    reduceGroups<BLOCK_SIZE><<<gridDim, blockDim>>>(dValsPtr, N);
    hipEventRecord(end);

    float ms;
    // Synchronizing to event. Event is last, same effect as hipDeviceSynchronize
    hipEventSynchronize(end);
    hipEventElapsedTime(&ms, start, end);
    std::cout << std::setw(20) << "Reduce Groups" << "\t" << ms << "ms \t" << mResult << std::endl;
}

__managed__ unsigned int mHappyNumSum;
__managed__ unsigned int mHappyNumCount;

__device__ bool isHappy(unsigned int num)
{
    while (num != 0 && num != 1 && num != 4)
    {
        unsigned int next_num = 0;
        for (unsigned int n = num; n > 0; n /= 10)
        {
            unsigned int t = n % 10;
            next_num += t * t;
        }
        num = next_num;
    }
    return num == 1;
}

__global__ void Sum10HappyNumbers(unsigned int N, unsigned int* mHappyNumbers)
{
    unsigned int input = cg::this_grid().thread_rank() + 1;

    bool happy = (input <= N) && isHappy(input);

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    auto g = cg::binary_partition(warp, happy);

    if (happy)
    {
        unsigned int offset;
        unsigned int partial_sum = cg::reduce(g, input, cg::plus<unsigned int>());
        if (g.thread_rank() == 0)
        {
            atomicAdd(&mHappyNumSum, partial_sum);
            offset = atomicAdd(&mHappyNumCount, g.size());
        }
        offset = g.shfl(offset, 0);
        mHappyNumbers[offset + g.thread_rank()] = input;
    }
}

void HappyNummbersWithGroups(unsigned int N)
{
    mHappyNumSum = 0;
    mHappyNumCount = 0;
    unsigned int* mHappyNumbers;
    hipMallocManaged((void**)&mHappyNumbers, sizeof(unsigned int) * N);

    Sum10HappyNumbers<<<(N + 255) / 256, 256>>>(N, mHappyNumbers);
    hipDeviceSynchronize();

    std::cout << "No. of happy numbers from 1 - " << N << ": " << mHappyNumCount << std::endl;
    std::cout << "Sum of happy numbers from 1 - " << N << ": " << mHappyNumSum << std::endl;
    std::cout << "\nList of happy numbers from 1 - " << N << ": ";

    std::sort(mHappyNumbers, mHappyNumbers + mHappyNumCount);
    for (int i = 0; i < mHappyNumCount; i++)
        std::cout << mHappyNumbers[i] << ((i == mHappyNumCount - 1) ? "\n" : ", ");
}

int main()
{
    std::cout << "==== Sample 16 - Cooperative Groups ====\n" << std::endl;
    /*
    Cooperative groups are very versatile. They can be created for entire
    grids, blocks, warps or opportunistically for converged threads. In
    essence, they package a range of recent CUDA features in an interface
    that abstracts away the low-level instructions, making CUDA code 
    easier to understand. As such, cooperative groups have a vast range
    of applications. The examples in this project cannot do them justice,
    for further use cases please consider the advanced NVIDIA Samples 
    that include detailed, elaborate applications.

    Expected output:
    1) Result of reduction, now computed with cooperative groups
    */

    //ReduceWithGroups();

    //RejectionSamplePiWithGroups();
    HappyNummbersWithGroups(1000);
    
    return 0;
}