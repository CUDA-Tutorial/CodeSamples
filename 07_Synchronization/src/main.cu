#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "../../shared/include/utility.h"

__global__ void WriteSlow(int* out, int val)
{
    samplesutil::WasteTime(1'000'000'000ULL);
    // Finally write value
    *out = val;
}

__global__ void Square(int* out)
{
    *out = *out * *out;
}

__global__ void ApproximatePi(bool synchronized)
{
    // Create block-shared variable for approximated Pi
    __shared__ float sPi;
    // Thread 0 computes Pi and stores it to shared memory
    if (threadIdx.x == 0)
        sPi = samplesutil::GregoryLeibniz(100'000);

    // Boolean decides whether threads synchronize or not
    if (synchronized)
        __syncthreads();

    // Every thread should now perform some task with Pi
    if (threadIdx.x%32 == 0)
        printf("Thread %d thinks Pi = %f\n", threadIdx.x, sPi);
}

int main()
{
    std::cout << "==== Sample 07 - Synchronization ====\n" << std::endl;
    /*
    Expected output:
        Demonstrating implicit synchronization:
        42 squared = 1764

        No __syncthreads after computing a block-shared Pi:
        Thread 32 thinks Pi = 0.000000
        Thread 64 thinks Pi = 0.000000
        Thread 96 thinks Pi = 0.000000
        Thread 0 thinks Pi = 3.141586
        (or similar. Results may be correct, but not safe!)

        __syncthreads after computing a block-shared Pi:
        Thread 64 thinks Pi = 3.141586
        Thread 96 thinks Pi = 3.141586
        Thread 0 thinks Pi = 3.141586
        Thread 32 thinks Pi = 3.141586
        (or similar)
    */

    /*
    Implicit synchronization between kernels and hipMemcpy:

    Consider the example below, where we have two kernels. The first
    kernel writes some data (slowly), the second modifies that data.
    Afterwards, we copy the modified data back to the CPU. By default,
    CUDA will assume that each command depends on the previous command
    and therefore will implicitly synchronize them: a kernel will only
    run when previous kernels have finished, note however that the CPU
    is free to continue working in the meantime. Similarly, hipMemcpy 
    will only start when all previous kernels have finished, but it 
    will also make the CPU wait until the copy has finished. Hence, we 
    don't need any other synchronization in this scenario. 
    */
    std::cout << "Demonstrating implicit synchronization:" << std::endl;
    // Allocate some device memory for kernels to work with
    int* dFooPtr;
    hipMalloc(&dFooPtr, sizeof(int));
    // First kernel sets device memory to 42 (slowly)
    WriteSlow<<<1,1>>>(dFooPtr, 42);
    // Second kernel squares value of variable
    Square<<<1,1>>>(dFooPtr);
    // Finally, we copy the result back to the CPU
    int foo;
    hipMemcpy(&foo, dFooPtr, sizeof(int), hipMemcpyDeviceToHost);
    // Print the result of the GPU's computation
    std::cout << "42 squared = " << foo << std::endl;

    /*
    Block-wide synchronization with syncthreads:

    The following kernels compute an approximation of Pi.
    The algorithm used is inherently sequential, therefore
    only one thread performs the communication and then 
    shares the result with all threads in the block. 
    However, while one thread is busy performing work, the
    other threads in the block are free to move ahead. 
    With __syncthreads, we force all threads in a block to 
    wait at a given point in the program until all other 
    threads get there.
    */
    std::cout << "\nNo __syncthreads after computing a block-shared Pi:" << std::endl;
    // Run once without syncthreads
    ApproximatePi<<<1, 128>>>(false);
    // Wait for printf to finish
    hipDeviceSynchronize();

    std::cout << "\n__syncthreads after computing a block-shared Pi:" << std::endl;
    // Run again with syncthreads
    ApproximatePi<<<1, 128>>>(true);
    // Wait for printf to finish
    hipDeviceSynchronize();

    return 0;
}

/*
Exercises:
1) Try launching a simple CUDA kernel 1000-10000 times in a loop, once
with hipDeviceSynchronize after each launch, once without it. 
What's the effect on runtime? Does anything change about the program behavior?
2) You can also memcpy from device to device. Perform a few of them (e.g. moving a
value from device location A to device location B to C) and then back to CPU and
confirm that everything happened properly in order.
3) Try running a kernel where the first 16 threads in each warp take one branch,
the other 16 take the other, with a syncthreads in each branch. What happens?
Why? Document what happens when the first 32 threads in a block of size 64 take 
one branch, the other 32 the other, with a syncthreads in each branch. What happens 
now? Provide your best guess why.
*/