#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include "../../shared/include/utility.h"

__device__ int dVal = 42;
__device__ int dOut;

// Very simple kernel that updates a variable
__global__ void CopyVal(const int* val)
{
	// Simulating a little work
	samplesutil::WasteTime(1'000'000ULL);
	// Update a global value
	dOut = *val;
}

void checkForErrors()
{
	// Catch errors that can be detected without synchronization, clear them
	hipError_t err;
	err = hipGetLastError();
	if (err == hipSuccess)
		std::cout << "hipGetLastError() before sync found no error" << std::endl;
	else
		std::cout << "hipGetLastError() before sync found error: " << hipGetErrorName(err) << ", CLEARS ERROR" << std::endl;

	// Catch errors that require explicit synchronization, do not clear them
	err = hipDeviceSynchronize();
	if (err == hipSuccess)
		std::cout << "hipDeviceSynchronize() found no error" << std::endl;
	else
		std::cout << "hipDeviceSynchronize() found error: " << hipGetErrorName(err) << ", KEEPS ERROR" << std::endl;

	// If errors were found via synchronization, hipGetLastError clears them
	err = hipGetLastError();
	if (err == hipSuccess)
		std::cout << "hipGetLastError() after sync found no error" << std::endl;
	else
		std::cout << "hipGetLastError() after sync found error: " << hipGetErrorName(err) << ", CLEARS ERROR" << std::endl;

	std::cout << std::endl;
}

#define PRINT_RUN_CHECK(S)		\
std::cout << #S << std::endl;	\
S;								\
checkForErrors();

int main()
{
	std::cout << "==== Sample 12 - Error Handling ====\n" << std::endl;
	/*
	 Many functions in the CUDA API return error codes that indicate
	 that something has gone wrong. However, this error is not 
	 necessarily caused by the function that returns it. Kernels and
	 asynchronous memcopies, e.g., return immediately and may only
	 encounter errors after the return value is observed on the CPU. 
	 Such errors can be detected at some later point, for instance by
	 a synchronous function like hipMemcpy or hipDeviceSynchronize,
	 or by hipGetLastError after a synchronization. To ensure that 
	 every single CUDA call worked without error, we would have to 
	 sacrifice concurrency and asynchronicity. Hence, error checking 
	 is, in practice, rather opportunistic and happens e.g. at runtime 
	 when an algorithm is synchronized anyway or when we debug misbehaving 
	 code. The error checking in this code is thus not practical and only 
	 serves to illustrate how different mechanisms detect previous errors. 

	 Expected output:

		(CopyVal<<<1, 1>>>(validDAddress))
		hipGetLastError() before sync found no error
		hipDeviceSynchronize() found no error
		hipGetLastError() after sync found no error

		(CopyVal<<<1, (1<<16)>>>(validDAddress))
		hipGetLastError() before sync found error: hipErrorInvalidConfiguration, CLEARS ERROR
		hipDeviceSynchronize() found no error
		hipGetLastError() after sync found no error

		(CopyVal<<<1, 1>>>(nullptr))
		hipGetLastError() before sync found no error
		hipDeviceSynchronize() found error: hipErrorIllegalAddress, KEEPS ERROR
		hipGetLastError() after sync found error: hipErrorIllegalAddress, CLEARS ERROR

		cudaErrorInvalidPc: invalid program counter
	*/

	int* validDAddress;
	// A function may return an error code - should check those for success
	hipError_t err = hipGetSymbolAddress((void**)&validDAddress, dVal);

	if (err != hipSuccess)
		// If an error occurred, identify it with hipGetErrorName and react!
		std::cout << hipGetErrorName(err) << std::endl;
	// Alternatively, you may peek at the last error to see if the program is ok
	err = hipPeekAtLastError();
	// Getting the last error effectively resets it. Useful after reacting to it
	err = hipGetLastError();

	/* 
	Launching a kernel with proper configuration and parameters.
	If the system is set up correctly, this should succeed.
	*/
	PRINT_RUN_CHECK((CopyVal<<<1, 1>>>(validDAddress)));

	/* 
	Launching a kernel with bigger block than possible.
	hipGetLastError() can catch SOME errors without synchronizing!
	*/
	PRINT_RUN_CHECK((CopyVal<<<1, (1<<16)>>>(validDAddress)));

	/*
	Launching a kernel with invalid address - error occurs after launch.
	hipGetLastError() alone may miss this without synchronization.
	*/
	PRINT_RUN_CHECK((CopyVal<<<1, 1>>>(nullptr)));

	// For any kind of error, CUDA also provides a more verbose description.
	std::cout << hipGetErrorName(cudaErrorInvalidPc) << ": " << hipGetErrorString(cudaErrorInvalidPc) << std::endl;
}

/*
Exercises:
1) Write a program that creates many pinned large allocations, and stop when 
the first error occurs. What is this error? When and why does it occur?
2) hipMemcpy can implicitly synchronize the GPU and CPU, hence its return values
can be used to find any errors of kernels that were launched before it. Demonstrate
this for a simple example where a kernel does something illegal that you discover
using hipMemcpy.
3) Try to produce an exotic error that does not occur already occur in this program
*/
