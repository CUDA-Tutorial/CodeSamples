#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>

// A simple kernel function to keep threads busy for a while
__global__ void busy()
{
	const int start = clock();
	while ((clock() - start) < 1'000'000'000);
	printf("I'm awake!\n");
}

void runTasksSequentially(unsigned int numTasks)
{
	// We use hipStreamPerThread here. It makes no difference
	// for the program flow because we are only single-threaded
	// anyway, but capturing the application-wide default stream 
	// hipStreamLegacy is not permitted by the graph API.

	for (int i = 0; i < numTasks; i++)
		busy<<<1, 1, 0, hipStreamPerThread>>>();
}

void runTasksWithStreams(unsigned int numTasks)
{
	/* 
	This stream-based function that can be directly captured with graph API.
	Events are used to encode dependencies / start / end of capture.
	*/
	std::vector<hipStream_t> streams(numTasks);
	std::vector<hipEvent_t> finished(numTasks);
	for (int i = 0; i < numTasks; i++)
	{
		hipStreamCreate(&streams[i]);
		hipEventCreate(&finished[i]);
	}	
	// We need an additional event to represent the capture start
	hipEvent_t start;
	hipEventCreate(&start);
	// Immediately record the starting event so other streams can connect to it
	hipEventRecord(start, hipStreamPerThread);
	// All other streams must connect to origin stream via event to get captured
	for (int i = 0; i < numTasks; i++)
	{
		// Establish dependency / connection to origin (is now included in capture)
		hipStreamWaitEvent(streams[i], start);
		// Run actual task (kernel) in stream
		busy<<<1, 1, 0, streams[i]>>>();
		// Record end event of this stream so origin can wait on it
		hipEventRecord(finished[i], streams[i]);
	}
	// Origin stream waits until all custom streams have finished their task
	for (int i = 0; i < numTasks; i++)
	{
		hipStreamWaitEvent(hipStreamPerThread, finished[i]);
		hipStreamDestroy(streams[i]);
	}
}

template <typename T, typename ...P>
hipGraphExec_t recordGraphFromFunction(const T& func, P ...params)
{
	hipGraph_t graph;
	hipGraphCreate(&graph, 0);

	hipStreamBeginCapture(hipStreamPerThread, hipStreamCaptureModeGlobal);
	func(params...);
	hipStreamEndCapture(hipStreamPerThread, &graph);

	hipGraphExec_t instance;
	hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
	hipGraphDestroy(graph);
	return instance;
}

hipGraphExec_t buildGraphForParallelTasks(unsigned int numTasks)
{
	hipGraph_t graph;
	hipGraphCreate(&graph, 0);

	std::vector<hipGraphNode_t> nodes(numTasks);
	hipKernelNodeParams params = { busy, {1,1,1}, {1,1,1}, 0, nullptr, nullptr };

	for (int i = 0; i < numTasks; i++)
		hipGraphAddKernelNode(&nodes[i], graph, nullptr, 0, &params);

	hipGraphExec_t instance;
	hipGraphInstantiate(&instance, graph, 0, 0, 0);
	hipGraphDestroy(graph);

	return instance;
}

int main()
{
	std::cout << "==== Sample 15 - Graph API ====\n" << std::endl;
	/*

	The CUDA graph does NOT include synchronization methods with the CPU!
	This means that waiting actions (e.g., until all streams have finished)
q	must be modelled via dependencies/events instead.
	*/

	constexpr int TASKS = 4;

	std::cout << "Launching multiple tasks sequentially" << std::endl;

	runTasksSequentially(TASKS);
	hipDeviceSynchronize();

	std::cout << "Running recorded graph from existing sequential code" << std::endl;

	hipGraphExec_t recordedSequential = recordGraphFromFunction(runTasksSequentially, TASKS);
	hipGraphLaunch(recordedSequential, 0);
	hipDeviceSynchronize();

	std::cout << "Launching multiple tasks with streams" << std::endl;

	runTasksWithStreams(TASKS);
	hipDeviceSynchronize();

	std::cout << "Running recorded graph from existing stream-based code" << std::endl;

	hipGraphExec_t recordedStreams = recordGraphFromFunction(runTasksWithStreams, TASKS);
	hipGraphLaunch(recordedStreams, 0);
	hipDeviceSynchronize();

	std::cout << "Running manually-built graph that behaves like streams" << std::endl;

	hipGraphExec_t instanceBuilt = buildGraphForParallelTasks(TASKS);
	hipGraphLaunch(instanceBuilt, 0);
	hipDeviceSynchronize();

	return 0;
}